/*
 * To solve this question, the basic flow is given as follows:
 * 1. Reading the JSON file and Map product IDs (asin) to integers
 * 2. Then, product_ids[] and ratings[] are created. Both these arrays are populated in CPU itself. No GPU involvement here!
 * 3. We will create a CUDA kernel with Input: product_ids[], ratings[], num_reviews and Output: total_rating_per_product[], count_per_product[]
 * 4. For each review, AtomicAdd to product's total rating and product's count
 * 5. After kernel: Average is computed, sorting happens and pick top 10
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX_REVIEWS 6800000
#define MAX_STR_LEN 15
#define HASH_TABLE_SIZE 100003
#define MAX_LINE_LENGTH 4096
#define MAX_PRODUCTS HASH_TABLE_SIZE

/*
 * In the code below, we are implementing a Set in C
 * This set stores the unique products along with their unique integer IDs which I give!
 */
typedef struct StringNode
{
    char str[MAX_STR_LEN];
    int id;
    struct StringNode *next;
} StringNode;

StringNode *hash_table[HASH_TABLE_SIZE];
int next_id = 0;

unsigned int hash_string(const char *str)
{
    unsigned int hash = 5381;

    while (*str)
    {
        hash = ((hash << 5) + hash) + *str++;
    }
    return hash % HASH_TABLE_SIZE;
}

int insert_string(const char *str)
{
    unsigned int index = hash_string(str);
    StringNode *curr = hash_table[index];

    while (curr)
    {
        if (strcmp(curr->str, str) == 0)
        {
            return curr->id; // already exists
        }

        curr = curr->next;
    }

    StringNode *new_node = (StringNode *)malloc(sizeof(StringNode));
    strcpy(new_node->str, str);
    new_node->id = next_id++;
    new_node->next = hash_table[index];
    hash_table[index] = new_node;

    return new_node->id;
}

void free_string_set()
{
    for (int i = 0; i < HASH_TABLE_SIZE; i++)
    {
        StringNode *curr = hash_table[i];
        while (curr)
        {
            StringNode *temp = curr;
            curr = curr->next;
            free(temp);
        }
    }
}

/*
 * The code below is for the kernel.
 */

__device__ float d_rating_sums[MAX_PRODUCTS];
__device__ int d_rating_counts[MAX_PRODUCTS];

__global__ void handle_reviews(int *d_product_ids, float *d_ratings)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < MAX_REVIEWS)
    {
        int product = d_product_ids[idx];
        float rating = d_ratings[idx];

        atomicAdd(&d_rating_sums[product], rating);
        atomicAdd(&d_rating_counts[product], 1);
    }
}

typedef struct
{
    int id;
    float avg_rating;
} ProductRating;

int compare(const void *a, const void *b)
{
    float diff = ((ProductRating *)b)->avg_rating - ((ProductRating *)a)->avg_rating;
    if (diff > 0)
        return 1;
    else if (diff < 0)
        return -1;
    else
        return 0;
}

int main()
{
    FILE *fp = fopen("Electronics_5.json", "r");
    int *product_ids = (int *)malloc(MAX_REVIEWS * sizeof(int));
    float *ratings = (float *)malloc(MAX_REVIEWS * sizeof(float));

    char line[MAX_LINE_LENGTH];
    long int line_num = 0;

    while (fgets(line, sizeof(line), fp))
    {
        char *token = strtok(line, ",");
        float rating = -1;
        char asin[MAX_STR_LEN] = "";

        while (token != NULL)
        {
            if (strstr(token, "\"overall\""))
            {
                char *colon = strchr(token, ':');
                if (colon)
                {
                    rating = atof(colon + 1);
                }
            }
            else if (strstr(token, "\"asin\""))
            {
                char *colon = strchr(token, ':');
                if (colon)
                {
                    char *value = colon + 1;
                    while (*value == ' ' || *value == '"')
                    {
                        value++;
                    }
                    char *end = value;
                    while (*end && *end != '"')
                    {
                        end++;
                    }
                    *end = '\0';
                    strcpy(asin, value);
                }
            }

            token = strtok(NULL, ",");
        }

        if (rating >= 0 && asin[0] != '\0')
        {
            int id = insert_string(asin);
            product_ids[line_num] = id;
            ratings[line_num] = rating;
            line_num++;
        }
    }

    char **id_to_asin = (char **)malloc(next_id * sizeof(char *));

    for (int i = 0; i < HASH_TABLE_SIZE; i++)
    {
        StringNode *curr = hash_table[i];
        while (curr)
        {
            id_to_asin[curr->id] = strdup(curr->str); // copy the asin string
            curr = curr->next;
        }
    }

    int *d_product_ids;
    float *d_ratings;

    hipMalloc(&d_product_ids, MAX_REVIEWS * sizeof(int));
    hipMalloc(&d_ratings, MAX_REVIEWS * sizeof(float));

    hipMemcpy(d_product_ids, product_ids, MAX_REVIEWS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ratings, ratings, MAX_REVIEWS * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    int blocks = (MAX_REVIEWS + threadsPerBlock - 1) / threadsPerBlock;

    handle_reviews<<<blocks, threadsPerBlock>>>(d_product_ids, d_ratings);
    hipDeviceSynchronize();

    float h_rating_sums[MAX_PRODUCTS];
    int h_rating_counts[MAX_PRODUCTS];

    hipMemcpyFromSymbol(h_rating_sums, HIP_SYMBOL(d_rating_sums), sizeof(h_rating_sums));
    hipMemcpyFromSymbol(h_rating_counts, HIP_SYMBOL(d_rating_counts), sizeof(h_rating_counts));

    for (long i = 0; i < MAX_PRODUCTS; i++)
    {
        if (h_rating_counts[i] > 0)
        {
            h_rating_sums[i] /= h_rating_counts[i];
        }
        else
        {
            h_rating_sums[i] = 0.0f; // or some sentinel value
        }
    }

    ProductRating *products = (ProductRating *)malloc(MAX_PRODUCTS * sizeof(ProductRating));
    int valid_products = 0;

    for (int i = 0; i < MAX_PRODUCTS; i++)
    {
        if (h_rating_counts[i] > 0)
        {
            products[valid_products].id = i;
            products[valid_products].avg_rating = h_rating_sums[i];
            valid_products++;
        }
    }

    qsort(products, valid_products, sizeof(ProductRating), compare);

    printf("\nTop 10 Products by Average Rating:\n");
    for (int i = 0; i < 10 && i < valid_products; i++)
    {
        int id = products[i].id;
        printf("ASIN: %s, Avg Rating: %.2f\n", id_to_asin[id], products[i].avg_rating);
    }

    return 0;
}